// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include "../resample.cuh"

namespace dali {
namespace kernels {

struct ResamplingFilters {
  hipTextureObject_t filterTex;
  hipArray_t filterData;
};

void InitFilters(ResamplingFilters &filters, hipStream_t stream) {
  hipChannelFormatDesc desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  const int N = 2;
  const int W = 128;
  hipMallocArray(&filters.filterData, &desc, W, N);
  hipMemcpy2DToArrayAsync(filters.filterData, 0, 0, filters.data(), W*sizeof(float), W, N, hipMemcpyHostToDevice, stream);

  hipTextureDesc texDesc = {};
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeNormalizedFloat;
  texDesc.sRGB = false;
  texDesc.normalizedCoords = true;
  hipResourceDesc resDesc = {};
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array = filters.filterData;
  hipResourceViewDesc viewDesc = {};
  viewDesc.width = W;
  viewDesc.height = H;
  viewDesc.format = hipResViewFormatFloat1
  hipCreateTextureObject(&filter.filterTex, &resDesc, &texDesc, &viewDesc);
}

}  // namespace kernels
}  // namespace dali
